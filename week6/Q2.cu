#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ void selection_sort(int *data, int left, int right) {
    for (int i = left; i <= right; ++i) {
        int min_val = data[i];
        int min_idx = i;

        for (int j = i + 1; j <= right; ++j) {
            int val_j = data[j];
            if (val_j < min_val) {
                min_idx = j;
                min_val = val_j;
            }
        }

        if (i != min_idx) {
            data[min_idx] = data[i];
            data[i] = min_val;
        }
    }
}

__global__ void selection_sort_kernel(int *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        selection_sort(data, idx, n - 1);
    }
}

void parallelSelectionSort(int *h_arr, int n) {
    int *d_arr;

    hipMalloc((void**)&d_arr, n * sizeof(int));

    hipMemcpy(d_arr, h_arr, n * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 1; 
    int numBlocks = n;  
    dim3 dimBlock(blockSize, 1, 1);
    dim3 dimGrid(numBlocks, 1, 1);

    selection_sort_kernel<<<dimGrid, dimBlock>>>(d_arr, n);
    hipDeviceSynchronize();
    hipMemcpy(h_arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

int main() {
    int h_arr[] = {5, 2, 9, 1, 5, 6};
    int n = sizeof(h_arr) / sizeof(h_arr[0]);

    parallelSelectionSort(h_arr, n);

    printf("Sorted array: ");
    for (int i = 0; i < n; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    return 0;
}
//Sorted array: 1 1 5 5 6 9 
