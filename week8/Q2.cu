#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define M 3 
#define N 3  
#define K 3  

__global__ void multiplyMatricesRowWise(int *A, int *B, int *C, int numColsA, int numColsB) {
    int row = blockIdx.x;  
    if (row < M) {
        for (int col = 0; col < numColsB; col++) {
            int sum = 0;
            for (int k = 0; k < numColsA; k++) {
                sum += A[row * numColsA + k] * B[k * numColsB + col];
            }
            C[row * numColsB + col] = sum;
        }
    }
}

__global__ void multiplyMatricesColWise(int *A, int *B, int *C, int numRowsA, int numColsB) {
    int col = blockIdx.x;
    if (col < N) {
        for (int row = 0; row < numRowsA; row++) {
            int sum = 0;
            for (int k = 0; k < K; k++) {
                sum += A[row * K + k] * B[k * N + col];
            }
            C[row * N + col] = sum;
        }
    }
}

__global__ void multiplyMatricesElementWise(int *A, int *B, int *C, int numColsA, int numColsB) {
    int row = blockIdx.x;
    int col = threadIdx.x;
    if (row < M && col < N) {
        int sum = 0;
        for (int k = 0; k < K; k++) {
            sum += A[row * numColsA + k] * B[k * numColsB + col];
        }
        C[row * numColsB + col] = sum;
    }
}

void printMatrix(int *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;

    size_t sizeA = M * K * sizeof(int);
    size_t sizeB = K * N * sizeof(int);
    size_t sizeC = M * N * sizeof(int);
    
    A = (int*)malloc(sizeA);
    B = (int*)malloc(sizeB);
    C = (int*)malloc(sizeC);

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            A[i * K + j] = i + j; 
        }
    }

    for (int i = 0; i < K; i++) {
        for (int j = 0; j < N; j++) {
            B[i * N + j] = i - j; 
        }
    }
    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);
    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);
    multiplyMatricesRowWise<<<M, 1>>>(d_A, d_B, d_C, K, N);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
    printf("Resultant Matrix (Approach a - Row-wise):\n");
    printMatrix(C, M, N);
    multiplyMatricesColWise<<<N, 1>>>(d_A, d_B, d_C, M, N);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
    printf("\nResultant Matrix (Approach b - Column-wise):\n");
    printMatrix(C, M, N);
    multiplyMatricesElementWise<<<M, N>>>(d_A, d_B, d_C, K, N);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
    printf("\nResultant Matrix (Approach c - Element-wise):\n");
    printMatrix(C, M, N);
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
