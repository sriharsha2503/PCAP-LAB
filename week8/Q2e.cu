#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define M 3 
#define N 3  
#define K 3  

__global__ void multiplyMatricesRowWise(int *A, int *B, int *C, int numColsA, int numColsB) {
    int row = blockIdx.x;  
    if (row < M) {
        for (int col = 0; col < numColsB; col++) {
            int sum = 0;
            for (int k = 0; k < numColsA; k++) {
                sum += A[row * numColsA + k] * B[k * numColsB + col];
            }
            C[row * numColsB + col] = sum;
        }
    }
}

__global__ void multiplyMatricesColWise(int *A, int *B, int *C, int numRowsA, int numColsB) {
    int col = blockIdx.x;
    if (col < N) {
        for (int row = 0; row < numRowsA; row++) {
            int sum = 0;
            for (int k = 0; k < K; k++) {
                sum += A[row * K + k] * B[k * N + col];
            }
            C[row * N + col] = sum;
        }
    }
}

__global__ void multiplyMatricesElementWise(int *A, int *B, int *C, int numColsA, int numColsB) {
    int row = blockIdx.x;
    int col = threadIdx.x;
    if (row < M && col < N) {
        int sum = 0;
        for (int k = 0; k < K; k++) {
            sum += A[row * numColsA + k] * B[k * numColsB + col];
        }
        C[row * numColsB + col] = sum;
    }
}

void printMatrix(int *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;

    size_t sizeA = M * K * sizeof(int);
    size_t sizeB = K * N * sizeof(int);
    size_t sizeC = M * N * sizeof(int);
    
    A = (int*)malloc(sizeA);
    B = (int*)malloc(sizeB);
    C = (int*)malloc(sizeC);

    // Taking input from the user for matrix A
    printf("Enter the elements of matrix A (3x3):\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            printf("A[%d][%d]: ", i, j);
            scanf("%d", &A[i * K + j]);
        }
    }

    // Taking input from the user for matrix B
    printf("Enter the elements of matrix B (3x3):\n");
    for (int i = 0; i < K; i++) {
        for (int j = 0; j < N; j++) {
            printf("B[%d][%d]: ", i, j);
            scanf("%d", &B[i * N + j]);
        }
    }

    // Allocating memory on the device
    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    // Copy data from host to device
    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    // Row-wise multiplication (Approach a)
    multiplyMatricesRowWise<<<M, 1>>>(d_A, d_B, d_C, K, N);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
    printf("Resultant Matrix (Approach a - Row-wise):\n");
    printMatrix(C, M, N);

    // Column-wise multiplication (Approach b)
    multiplyMatricesColWise<<<N, 1>>>(d_A, d_B, d_C, M, N);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
    printf("\nResultant Matrix (Approach b - Column-wise):\n");
    printMatrix(C, M, N);

    // Element-wise multiplication (Approach c)
    multiplyMatricesElementWise<<<M, N>>>(d_A, d_B, d_C, K, N);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
    printf("\nResultant Matrix (Approach c - Element-wise):\n");
    printMatrix(C, M, N);

    // Free allocated memory
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

