#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#define N 1024  

__global__ void countWordKernel(const char *sentence, const char *word, unsigned int *d_count, int sentence_len, int word_len) 
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx <= sentence_len - word_len) 
    {
        bool match = true;
        for (int i = 0; i < word_len; ++i) 
        {
            if (sentence[idx + i] != word[i]) 
            {
                match = false;
                break;
            }
        }
        if (match) 
        {
            atomicAdd(d_count, 1);
        }
    }
}

void checkCudaError(hipError_t error, const char* msg) 
{
    if (error != hipSuccess) 
    {
        printf("CUDA error (%s): %s\n", msg, hipGetErrorString(error));
        exit(-1);
    }
}

int main() {
    char sentence[N];
    char word[50];
    char *d_sentence, *d_word;
    unsigned int *d_count;
    unsigned int *count;  
    count = (unsigned int*)malloc(sizeof(unsigned int));
    *count = 0; 
    printf("Enter a sentence: ");
    fgets(sentence, N, stdin);  
    printf("Enter the word to search for: ");
    fgets(word, 50, stdin); 
    sentence[strcspn(sentence, "\n")] = 0;
    word[strcspn(word, "\n")] = 0;
    
    int sentence_len = strlen(sentence);
    int word_len = strlen(word);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    checkCudaError(hipMalloc((void**)&d_sentence, (sentence_len + 1) * sizeof(char)), "Allocating memory for d_sentence");
    checkCudaError(hipMalloc((void**)&d_word, (word_len + 1) * sizeof(char)), "Allocating memory for d_word");
    checkCudaError(hipMalloc((void**)&d_count, sizeof(unsigned int)), "Allocating memory for d_count");
    checkCudaError(hipMemcpy(d_sentence, sentence, (sentence_len + 1) * sizeof(char), hipMemcpyHostToDevice), "Copying sentence to device");
    checkCudaError(hipMemcpy(d_word, word, (word_len + 1) * sizeof(char), hipMemcpyHostToDevice), "Copying word to device");
    checkCudaError(hipMemcpy(d_count, count, sizeof(unsigned int), hipMemcpyHostToDevice), "Copying initial count to device");
    dim3 dimGrid((sentence_len + 256 - 1) / 256, 1, 1);
    dim3 dimBlock(256, 1, 1);
    countWordKernel<<<dimGrid, dimBlock>>>(d_sentence, d_word, d_count, sentence_len, word_len);
    checkCudaError(hipGetLastError(), "Kernel launch failed");
    checkCudaError(hipDeviceSynchronize(), "CUDA device synchronization failed");
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);   
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);    
    checkCudaError(hipMemcpy(count, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost), "Copying count back to host");
    printf("Total occurrences of the word '%s': %u\n", word, *count);
    printf("Time taken: %f milliseconds\n", elapsedTime);
    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_count);
    free(count); 
    return 0;
}

//Enter a sentence: i am agood person with very good 
//Enter the word to search for: good
//Total occurrences of the word 'good': 2
//Time taken: 1.804288 milliseconds
