#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#define MAX_LEN 1024
__global__ void genPatterns(const char *in, char *out, int len, int num) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < num) {
        int sub_len = len - idx;
        for (int i = 0; i < sub_len; ++i) {
            out[idx * len + i] = in[i];
        }
    }
}
int main() {
    char in[MAX_LEN];
    char *out;
    char *d_in, *d_out;
    printf("Enter string: ");
    fgets(in, MAX_LEN, stdin);
    in[strcspn(in, "\n")] = 0;
    int len = strlen(in);
    int num = len;
    hipMalloc((void**)&d_in, (len + 1) * sizeof(char));
    hipMalloc((void**)&d_out, (len * num + 1) * sizeof(char));
    hipMemcpy(d_in, in, (len + 1) * sizeof(char), hipMemcpyHostToDevice);
    int blkSize = 256;
    int blkCount = (num + blkSize - 1) / blkSize;
    genPatterns<<<blkCount, blkSize>>>(d_in, d_out, len, num);
    hipDeviceSynchronize();
    out = (char *)malloc(len * num * sizeof(char));
    hipMemcpy(out, d_out, (len * num + 1) * sizeof(char), hipMemcpyDeviceToHost);
    printf("Input: %s\n", in);
    printf("Pattern:");
    for (int i = 0; i < num; ++i) {
        int sub_len = len - i;
        printf("%.*s", sub_len, &out[i * len]);
    }
    printf("\n");
    hipFree(d_in);
    hipFree(d_out);
    free(out);
    return 0;
}
//Enter string: paragraph
//Input: paragraph
//Pattern:paragraphparagrapparagraparagrparagparaparpap
