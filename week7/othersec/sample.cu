#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#define N 1024

__global__ void CUDACount(char* A, unsigned int *d_count, int length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < length) {
        if (A[i] == 'a') {
            atomicAdd(d_count, 1);
        }
    }
}

int main() {
    char A[N];
    char *d_A;
    unsigned int *d_count, count = 0, *result;

    // Take user input
    printf("Enter a string: ");
    fgets(A, N, stdin);  // Using fgets instead of gets for safety
    A[strcspn(A, "\n")] = 0;  // Remove newline character from fgets input

    int length = strlen(A);

    // Allocate memory on the device
    hipMalloc((void**)&d_A, length * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(unsigned int));
    hipHostMalloc((void**)&result, sizeof(unsigned int), hipHostMallocDefault);

    // Initialize count to 0 on the host
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

    // Copy input string to the device
    hipMemcpy(d_A, A, length * sizeof(char), hipMemcpyHostToDevice);

    // CUDA Event setup for measuring time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Launch kernel with enough threads to cover the entire string
    int threads_per_block = 256;
    int blocks_per_grid = (length + threads_per_block - 1) / threads_per_block;
    CUDACount<<<blocks_per_grid, threads_per_block>>>(d_A, d_count, length);

    // Error checking
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error));
    }

    // Wait for kernel to finish and record stop time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Copy the result back to the host
    hipMemcpy(result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Measure elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Print result
    printf("Total occurrences of 'a': %u\n", *result);
    printf("Time Taken: %f ms\n", elapsedTime);

    // Free device memory
    hipFree(d_A);
    hipFree(d_count);
    hipHostFree(result);

    return 0;
}
