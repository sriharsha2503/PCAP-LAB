#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void linearAlgebraOperation(float *x, float *y, float alpha, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Global thread index

    if (idx < N) {
        y[idx] = alpha * x[idx] + y[idx];  // Perform the operation y = alpha * x + y
    }
}

int main(void) {
    int N = 1000;  // Size of the vectors
    float alpha = 2.0f;  // Scalar value
    int size = N * sizeof(float);

    float *h_x = (float *)malloc(size);
    float *h_y = (float *)malloc(size);
    float *d_x, *d_y;

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_x[i] = (float)(i);  // x = 0, 1, 2, 3, ...
        h_y[i] = (float)(i * 2);  // y = 0, 2, 4, 6, ...
    }

    // Allocate device memory
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);

    // Copy data from host to device
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

    // Define block size and number of blocks
    int THREADS_PER_BLOCK = 256;
    int numBlocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch the kernel
    linearAlgebraOperation<<<numBlocks, THREADS_PER_BLOCK>>>(d_x, d_y, alpha, N);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

    // Print the first 10 results for verification
    printf("First 10 results of y = alpha * x + y:\n");
    for (int i = 0; i < (N < 10 ? N : 10); i++) {
        printf("y[%d] = %f\n", i, h_y[i]);
    }

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);

    // Free host memory
    free(h_x);
    free(h_y);

    return 0;
}

//nvcc additional1.cu -o ad1
//./ad1
//First 10 results of y = alpha * x + y:
//y[0] = 0.000000
//y[1] = 4.000000
//y[2] = 8.000000
//y[3] = 12.000000
//y[4] = 16.000000
//y[5] = 20.000000
//y[6] = 24.000000
//y[7] = 28.000000
//y[8] = 32.000000
//y[9] = 36.000000

