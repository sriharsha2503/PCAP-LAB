#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define THREADS_PER_BLOCK 256

__global__ void addVectors(int* A, int* B, int* C, int N) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void addVectors_BlockSizeAsN(int* A, int* B,int* C,int N)
{ int idx=threadIdx.x;
  if(idx<N)
  {
   C[idx]=A[idx]+B[idx];
  }
}  

__global__ void addVectors_NThreads(int* A,int* B,int* C,int N)
{
  int idx=blockIdx.x;
  if(idx<N)
  {
   C[idx]=A[idx]+B[idx];
  }
}

int main()
{
 int N=1024;
 int *A,*B,*C;
 int *d_A,*d_B,*d_C;
 
 A=(int*)malloc(N*sizeof(int));
 B=(int*)malloc(N*sizeof(int));
 C=(int*)malloc(N*sizeof(int));
 
 for(int i=0;i<N;i++)
 {
  A[i]=i+1;
  B[i]=(i+1)*2;
 }
 
 hipMalloc((void**)&d_A,N*sizeof(int));
 hipMalloc((void**)&d_B,N*sizeof(int));
 hipMalloc((void**)&d_C,N*sizeof(int));
 
 hipMemcpy(d_A,A,N*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(d_B,B,N*sizeof(int),hipMemcpyHostToDevice);
 addVectors_BlockSizeAsN<<<1,N>>>(d_A,d_B,d_C,N);//1st kernel call
 hipDeviceSynchronize();
 hipMemcpy(C, d_C, N * sizeof(int), hipMemcpyDeviceToHost);
 printf("Result (Block size N): ");
 for (int i = 0; i < N; i++)
 {
   printf("%d ", C[i]);
 }
 printf("\n");
    
 addVectors_NThreads<<<N,1>>>(d_A,d_B,d_C,N);//2nd kernel call
 hipDeviceSynchronize();
 hipMemcpy(C, d_C, N * sizeof(int), hipMemcpyDeviceToHost);
 printf("Result (N Threads): ");
 for (int i = 0; i < N; i++)
 {
   printf("%d ", C[i]);
 }
 printf("\n");
 

  dim3 dimBlock(256, 1, 1);  
  dim3 dimGrid(ceil(N / 256.0), 1, 1); 
  addVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);//3rd kernel call 
  hipDeviceSynchronize();
  hipMemcpy(C, d_C, N * sizeof(int), hipMemcpyDeviceToHost);
  printf("Result for normal vector addition: ");
  for (int i = 0; i < N; i++) 
  {
    printf("%d ", C[i]);
  }
  printf("\n");


 free(A);
 free(B);
 free(C);
 hipFree(d_A);
 hipFree(d_B);
 hipFree(d_C);
 return 0;
}
